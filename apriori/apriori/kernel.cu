#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "CUDABackground.h"
#include <atomic>
#include <iostream>
#include <pplinterface.h>
#include "FileReader2.h"
#include "UserTableReader.h"
using namespace std;

__global__ void computeAverageType2(float**mainArray, unsigned short *mainArrayColumns, unsigned short *mainArrayRows)
{
	short column = blockIdx.x * blockDim.x + threadIdx.x + 1;
	float cur;
	if (column < *mainArrayColumns)
	{
		double total = 0;
		unsigned short count = 0;
		for (short i = 1; i < *mainArrayRows; i++)
		{
			cur = mainArray[i][column];
			if (cur >= 0 && cur <= 5)
			{
				total += cur;
				count++;
			}
		}
		mainArray[0][column] = total / count;
		//printf("%d, %d, %f, %d, %f\n", row, 0, total, count, mainArray[row][0]);
	}
}

__global__ void computeSimularMoviesType2(float**userArray, unsigned short *userArrayRows, float**movieArray, unsigned short *movieArrayColumns)
{
	short movie1 = blockDim.x * blockIdx.x + threadIdx.x + 1;
	short movie2 = blockDim.y * blockIdx.y + threadIdx.y + 1;
	if (movie1 < *movieArrayColumns && movie2 < *movieArrayColumns && movie1 >= movie2)
	{
		//printf("%d,%d\n", movie1, movie2);

		double top = 0;
		float topLeft = 0;
		float topRight = 0;
		double bottomLeft = 0;
		double bottomRight = 0;

		for (short i = 1; i < *userArrayRows; i++)	//for every user
		{
			topLeft = userArray[i][movie1];			//get user rating for movie 1
			if (topLeft < 0 || topLeft > 5)			//if its not filled out by user, set to 0
				topLeft = 0;
			else
			{
				topLeft -= userArray[0][movie1];	//subtracting the average for that movie
			}

			topRight = userArray[i][movie2]; 		//get user rating for movie 2
			if (topRight < 0 || topRight > 5)		//if its not filled out by user, set to 0
				topRight = 0;
			else
			{
				topRight -= userArray[0][movie2]; //subtracting the average for that movie
			}							

			top += topRight * topLeft;				//compute this one and add to sum

			bottomLeft += topLeft * topLeft;		//A^2 and add to A's sum
			bottomRight += topRight * topRight;		//B^2 and add to B's sum
		}

		if (movie1 < 11 && movie2 < 11)
			printf("(%d,%d): \t%lf, %lf, %lf %lf\n", movie1, movie2, sqrt(bottomLeft), sqrt(bottomRight), top, top / (sqrt(bottomLeft) * sqrt(bottomRight)));

		movieArray[movie1][movie2] = movieArray[movie2][movie1] = top / (sqrt(bottomLeft) * sqrt(bottomRight));
		if (movie1 < 11 && movie2 < 11)
			printf("\t%d,%d: \t%lf, %lf, %lf %lf\n", movie1, movie2, bottomLeft, bottomRight, top, movieArray[movie2][movie1]);

	}

}




__global__ void computeAverage(float**mainArray, unsigned short *mainArrayColumns, unsigned short *mainArrayRows)
{
	short row = blockIdx.x * blockDim.x + threadIdx.x+1;
	float cur;
	if (row < *mainArrayRows)
	{
		double total = 0;
		unsigned short count = 0;
		for (short i = 1; i < *mainArrayColumns; i++)
		{
			cur = mainArray[row][i];
			if (cur >= 0 && cur <= 5)
			{
				total += mainArray[row][i];
				count++;
			}
		}
		mainArray[row][0] = total / count;
		//printf("%d, %d, %f, %d, %f\n", row, 0, total, count, mainArray[row][0]);
	}
}

__global__ void computeRecommendedMovies(float**userArray, unsigned short *userArrayColumns, unsigned short *userArrayRows, float**movieArray, bool **didSelect)
{
	short movie = blockDim.x * blockIdx.x + threadIdx.x + 1;
	short user = blockDim.y * blockIdx.y + threadIdx.y + 1;
	float tempSim;
	short selected = 0;
	float top5[6];
	short top5Index[6];
	if (movie < *userArrayColumns && user < *userArrayRows)
	{
		for (int i = 1; i < *userArrayColumns; i++)
		{
			if (didSelect[movie][i])
			{
				tempSim = movieArray[movie][i];
				if (selected < 5)
				{
					top5[selected] = tempSim;
					top5Index[selected] = i;
					selected++;
				}
				else
				{
					top5[0] = tempSim;
					top5Index[0] = i;
					float temp;
					short temp2;
					for (int i2 = 0; i2 <= 5; i2++)
					{
						for (int j = 0; j < 5; j++)
						{
							if (top5[j] > top5[j + 1])
							{
								temp = top5[j];
								temp2 = top5Index[j];
								top5[j] = top5[j + 1];
								top5Index[j] = top5Index[j + 1];
								top5[j + 1] = temp;
								top5Index[j + 1] = temp2;
							}
							else if (top5[j] == top5[j + 1] && top5Index[j] > top5Index[j+1])
							{
								temp = top5[j];
								temp2 = top5Index[j];
								top5[j] = top5[j + 1];
								top5Index[j] = top5Index[j + 1];
								top5[j + 1] = temp;
								top5Index[j + 1] = temp2;
							}
						}
					}
				}
			}
		}
		double sum;
		for (int i = 1; i <=5; i++)
			sum+= top5[i] * userArray[user][top5Index[i]];
		userArray[user][movie] = sum / selected;
	}

}


__global__ void computeSimularMovies(float**userArray, unsigned short *userArrayRows, float**movieArray, unsigned short *movieArrayColumns)
{
	short movie1 = blockDim.x * blockIdx.x + threadIdx.x + 1;
	short movie2 = blockDim.y * blockIdx.y + threadIdx.y + 1;
	if (movie1 < *movieArrayColumns && movie2 < *movieArrayColumns && movie1 >= movie2)
	{
		//printf("%d,%d\n", movie1, movie2);
		
		double top = 0;
		float topLeft = 0;
		float topRight = 0;
		double bottomLeft = 0;
		double bottomRight = 0;

		for (short i = 1; i < *userArrayRows; i++)	//for every user
		{
			topLeft = userArray[i][movie1];			//get user rating for movie 1
			if (topLeft < 0 || topLeft > 5)			//if its not filled out by user, set to 0
				topLeft = 0;
			else
			{
				topLeft -= userArray[i][0]; //subtracting the average for that user
				if (movie1 == 2 && movie2 == 1)
					printf("\ntopLeft: %lf", topLeft);
			}				

			topRight = userArray[i][movie2]; 		//get user rating for movie 2
			if (topRight < 0 || topRight > 5)		//if its not filled out by user, set to 0
				topRight = 0;	
			else
			{
				topRight -= userArray[i][0]; //subtracting the average for that user
				if (movie1 == 2 && movie2 == 1)
					printf("\ntopRight: %lf", topRight);
			}							//subtracting the average for that user

			top += topRight * topLeft;				//compute this one and add to sum

			bottomLeft += topLeft * topLeft;		//A^2 and add to A's sum
			bottomRight += topRight * topRight;		//B^2 and add to B's sum
		}

		if (movie1 < 11 && movie2 < 11)
			printf("(%d,%d): \t%lf, %lf, %lf %lf\n", movie1, movie2, sqrt(bottomLeft), sqrt(bottomRight), top, top / (sqrt(bottomLeft) * sqrt(bottomRight)));

		movieArray[movie1][movie2] = movieArray[movie2][movie1] = top / (sqrt(bottomLeft) * sqrt(bottomRight));
		if (movie1 < 11 && movie2 < 11)
			printf("\t%d,%d: \t%lf, %lf, %lf %lf\n", movie1, movie2, bottomLeft, bottomRight, top, movieArray[movie2][movie1]);

	}

}

float ** createArr(int size)
{
	float ** arr = (float**)malloc(sizeof(float*) * size);
	for(int i = 0; i < size+1; i++)
	{
		arr[i] = (float*)malloc(sizeof(float) * size + 1);
		for (int j = 0; j < size + 1; j++)
		{
			arr[i][j] = 0;
		}
	}
	return arr;
}

float ** createBlankUserMatrix(UserTableReader r, int columnMax)
{
	float ** arr = (float**)malloc(sizeof(float*) * columnMax+1);

	for (int i = 0; i < r.users.size() +1; i++)
	{
		arr[i] = (float*)malloc(sizeof(float) * columnMax + 1);
		for (int j = 0; j < r.users.size() + 1; j++)
		{
			arr[i][j] = 6.0;
		}
	}
	return arr;
}

bool ** createBlankUserDidReviewMatrix(UserTableReader r, int columnMax)
{
	bool ** arr = (bool**)malloc(sizeof(bool*) * columnMax + 1);

	for (int i = 0; i < r.users.size() +1; i++)
	{
		arr[i] = (bool*)malloc(sizeof(bool) * columnMax + 1);
		for (int j = 0; j < r.users.size() + 1; j++)
		{
			arr[i][j] = false;
		}
	}
	return arr;
}

void populateUserReviewMatrix(float **userReviewMatrix, bool **originalReviewMatrix, UserTableReader r, MovieReader m)
{
	auto vec = r.users;
	for (auto it = vec.begin(); it != vec.end(); ++it)
	{
		for (auto sit = (*it).ratedMovies.begin(); sit != (*it).ratedMovies.end(); ++sit)
		{
			userReviewMatrix[(*it).userID][m.movieIDMapper[(*sit).movieID]] = (*sit).rating;
			originalReviewMatrix[(*it).userID][m.movieIDMapper[(*sit).movieID]] = true;
		}
	}
}

hipError_t doAlgo()
{
	//int cudaCores = cuda.calculateCores();
	MovieReader m = MovieReader("movie.csv");
	UserTableReader r = UserTableReader("ratings.csv");
	float ** movieMatrix = createArr(m.movieCount);
	float ** userReviewMatrix = createBlankUserMatrix(r, m.movieCount);
	bool ** originalReviewMatrix = createBlankUserDidReviewMatrix(r, m.movieCount);
	populateUserReviewMatrix(userReviewMatrix, originalReviewMatrix, r, m);
	printf("%f, %f, %lf, %lf\n%lf, %lf, %lf, %lf\n", movieMatrix[1][1], movieMatrix[1][2], movieMatrix[1][3], movieMatrix[1][4], movieMatrix[2][1], movieMatrix[2][2], movieMatrix[2][3], movieMatrix[2][4]);


	float ** d_movieMatrix;
	float ** d_userReviewMatrix;
	bool ** d_didReviewMatrix;

	hipError_t cudaStatus;
	int movieMatrixColumns = m.movieCount + 1;
	int userReviewColumns = m.movieCount + 1;
	int userReviewRows = r.users.size() + 1;

	unsigned short * d_userReviewMatrixColumns;
	hipMalloc((void**)&d_userReviewMatrixColumns, sizeof(unsigned short) * 1);
	cudaStatus = hipMemcpy(d_userReviewMatrixColumns,&userReviewColumns,sizeof(unsigned short), hipMemcpyHostToDevice);


	unsigned short * d_userReviewMatrixRows;
	hipMalloc((void**)&d_userReviewMatrixRows, sizeof(unsigned short) * 1);
	cudaStatus = hipMemcpy(d_userReviewMatrixRows, &userReviewRows, sizeof(unsigned short), hipMemcpyHostToDevice);


	cudaStatus = hipMalloc((void***)&d_userReviewMatrix, userReviewRows * sizeof(float*));
	for (int i = 0; i < userReviewRows; i++)
	{
		float * temp;
		hipMalloc((void**) &(temp), sizeof(float)*userReviewColumns);
		hipMemcpy(temp, userReviewMatrix[i], sizeof(float) * userReviewColumns, hipMemcpyHostToDevice);
		hipMemcpy(d_userReviewMatrix + i, &temp, sizeof(float*), hipMemcpyHostToDevice); 
	}

	cudaStatus = hipMalloc((void***)&d_movieMatrix, movieMatrixColumns * sizeof(float*));
	for (int i = 0; i < movieMatrixColumns; i++)
	{
		float * temp;
		hipMalloc((void**) &(temp), sizeof(float)*movieMatrixColumns);
		hipMemcpy(temp, movieMatrix[i], sizeof(float) * movieMatrixColumns, hipMemcpyHostToDevice);
		hipMemcpy(d_movieMatrix + i, &temp, sizeof(float*), hipMemcpyHostToDevice);
	}

	cudaStatus = hipMalloc((void***)&d_didReviewMatrix, movieMatrixColumns * sizeof(bool*));
	for (int i = 0; i < userReviewRows; i++)
	{
		bool * temp;
		hipMalloc((void**) &(temp), sizeof(bool)*userReviewColumns);
		hipMemcpy(temp, originalReviewMatrix[i], sizeof(bool) * userReviewColumns, hipMemcpyHostToDevice);
		hipMemcpy(d_didReviewMatrix + i, &temp, sizeof(bool*), hipMemcpyHostToDevice);
	}



	int blockX = ceil(userReviewRows / 256.0);
	int blockY = ceil(userReviewRows / 16.0);
	int blockXType2 = ceil(userReviewColumns / 256);



	computeAverageType2 << <blockXType2, 256 >> > (d_userReviewMatrix, d_userReviewMatrixColumns, d_userReviewMatrixRows);
	printf("SUCCESS");
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	blockX = ceil(movieMatrixColumns / 16.0);
	blockY = ceil(movieMatrixColumns / 16.0);


	computeSimularMoviesType2<<<dim3(blockX, blockY), dim3(16,16) >>>(d_userReviewMatrix, d_userReviewMatrixRows, d_movieMatrix, d_userReviewMatrixColumns);
	cudaStatus = hipGetLastError();
	if (hipSuccess != hipGetLastError())
		printf("Error!\n");
	cudaStatus = hipGetLastError();
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	blockX = ceil(movieMatrixColumns / 16.0);
	blockY = ceil(userReviewRows / 16.0);
	computeRecommendedMovies<<<dim3(blockX, blockY), dim3(16, 16) >>>(d_userReviewMatrix, d_userReviewMatrixColumns, d_userReviewMatrixRows, d_movieMatrix, d_didReviewMatrix);
	cudaStatus = hipGetLastError();
	if (hipSuccess != hipGetLastError())
		printf("Error!\n");

	cudaStatus = hipGetLastError();
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}


Error:
	for (int i = 0; i < movieMatrixColumns; i++)
	{
		hipFree(d_movieMatrix+i);
	}
	for (int i = 0; i < userReviewRows; i++)
	{
		hipFree(d_userReviewMatrix + i);
		hipFree(d_didReviewMatrix + i);
	}
	hipFree(d_didReviewMatrix);
	hipFree(d_movieMatrix);
	hipFree(d_userReviewMatrix);
	hipFree(d_userReviewMatrixColumns);
	hipFree(d_userReviewMatrixRows);

	return cudaStatus;
}



int main()
{

	CUDABackground cuda = CUDABackground();
	doAlgo();
}