#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "CUDABackground.h"
#include "FileReader.h"
#include <atomic>
#include <iostream>
#include <pplinterface.h>
#include "FileReader2.h"
#include "UserTableReader.h"
using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t findFrequents(int**main, unsigned int *counts, int mainSize, int countsSize);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void findFrequents(int**mainArray, unsigned int *counts)
{
	int i = threadIdx.x;
	//int end = mainArray[i][0] + 1;
	printf("%d:\t%d\n", threadIdx.x, mainArray[i][1]);
	int * startAddy = mainArray[i];
	for(int j = 1; j <= 1; j++)
	{
		//atomicAdd(counts + mainArray[i][j], 1);
	}
}

short ** createArr(int size)
{
	short ** arr = (short**)malloc(sizeof(short*) * size);
	for(int i = 0; i < size; i++)
	{
		arr[i] = (short*)malloc(sizeof(short) * size + 1);
	}
	return arr;
}

short ** createBlankUserMatrix(UserTableReader r, int columnMax)
{
	short ** arr = (short**)malloc(sizeof(short*) * columnMax+1);

	for (int i = 0; i < r.users.size(); i++)
	{
		arr[i] = (short*)malloc(sizeof(short) * columnMax + 1);
	}
	return arr;
}

bool ** createBlankUserDidReviewMatrix(UserTableReader r, int columnMax)
{
	bool ** arr = (bool**)malloc(sizeof(bool*) * columnMax + 1);

	for (int i = 0; i < r.users.size(); i++)
	{
		arr[i] = (bool*)malloc(sizeof(bool) * columnMax + 1);
	}
	return arr;
}

void populateUserReviewMatrix(short **userReviewMatrix, bool **originalReviewMatrix, UserTableReader r, MovieReader m)
{
	auto vec = r.users;
	for (auto it = vec.begin(); it != vec.end(); ++it)
	{
		for (auto sit = (*it).ratedMovies.begin(); sit != (*it).ratedMovies.end(); ++sit)
		{
			userReviewMatrix[(*it).userID][m.movieIDMapper[(*sit).movieID]] = (*sit).rating;
		}
	}
}

void doAlgo()
{
	MovieReader m = MovieReader("movie.csv");
	UserTableReader r = UserTableReader("ratings.csv");
	short ** movieMatrix = createArr(m.movieCount);
	short ** userReviewMatrix = createBlankUserMatrix(r, m.movieCount);
	bool ** originalReviewMatrix = createBlankUserDidReviewMatrix(r, m.movieCount);
	populateUserReviewMatrix(userReviewMatrix, originalReviewMatrix, r, m);
	short ** d_movieMatrix;
	short * devptr;
	size_t pitch;
	hipError_t cudaStatus;


	cudaStatus = hipMalloc((void**)&d_movieMatrix, m.movieCount * sizeof(short*));
	hipMallocPitch(&devptr, &pitch, (m.movieCount + 1) * sizeof(short), (m.movieCount + 1));
	hipMemcpy2D(d_movieMatrix, pitch, movieMatrix, (m.movieCount + 1) * sizeof(short), (m.movieCount + 1) * sizeof(short), m.movieCount + 1, hipMemcpyHostToDevice);
 
	//short** temp_d_ptrs = (short **)malloc(sizeof(short*) * mainSize);
	for (int i = 0; i < m.movieCount; i++)
	{
		hipMalloc((void**)&temp_d_ptrs[i], sizeof(int)* (main[i][0] + 1)); // allocate for 1 int in each int pointer
		//hipMemcpy(temp, main[i], sizeof(int) * getsize, hipMemcpyHostToDevice); // copy data
		//hipMemcpy(devMain + i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}


Error:
	hipFree();
	hipFree();

	return cudaStatus;



}

hipError_t mallocCuda()
{
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	int** temp_d_ptrs = (int **)malloc(sizeof(int*) * mainSize);
	for (int i = 0; i < m; i++)
	{
		hipMalloc((void**)&temp_d_ptrs[i], sizeof(int)* (main[i][0] + 1)); // allocate for 1 int in each int pointer
		//hipMemcpy(temp, main[i], sizeof(int) * getsize, hipMemcpyHostToDevice); // copy data
		//hipMemcpy(devMain + i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}


Error:
	hipFree();
	hipFree();

	return cudaStatus;

}


int main()
{

	CUDABackground cuda = CUDABackground();
	doAlgo();
	FileReader f = FileReader("data.txt");






















	int cudaCores = cuda.calculateCores();
	unsigned int * frequency = (unsigned int *)malloc(sizeof(unsigned int) * f.maxNumber);
	for (int i = 0; i < f.maxNumber; i++)
	{
		frequency[i] = 0;
	}
	for (int i = 0; i < 50; i++)
	{
		printf("\n%d: ", i);
		for (int j = 0; j < f.master[i][0];j++)
			printf("%d,", f.master[i][j]);
	}
	cout<<frequency[0];
	1 + 1;
	hipError_t cudaStatus = findFrequents(f.master, frequency, f.count, f.maxNumber);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	int test2 = frequency[0];




    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t findFrequents(int* main[], unsigned int *counts, int mainSize, int countsSize)
{
	int **devMain = 0;
	unsigned int * devCounts = 0;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	int** temp_d_ptrs = (int **)malloc(sizeof(int*) * mainSize);
	for (int i = 0; i < mainSize; i++)
	{
		hipMalloc((void**)&temp_d_ptrs[i], sizeof(int)* (main[i][0] + 1)); // allocate for 1 int in each int pointer
		//hipMemcpy(temp, main[i], sizeof(int) * getsize, hipMemcpyHostToDevice); // copy data
		//hipMemcpy(devMain + i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMalloc((void**)&devCounts, countsSize * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devCounts, counts, countsSize * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	findFrequents << <1, 500 >> > (devMain, devCounts);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(counts, devCounts, countsSize * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devMain);
	hipFree(counts);

	return cudaStatus;

}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
