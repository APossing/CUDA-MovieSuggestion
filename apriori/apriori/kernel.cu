#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "CUDABackground.h"
#include <atomic>
#include <iostream>
#include <pplinterface.h>
#include "FileReader2.h"
#include "UserTableReader.h"
using namespace std;
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t findFrequents(int**main, unsigned int *counts, int mainSize, int countsSize);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void findFrequents(int**mainArray, unsigned int *counts)
{
	int i = threadIdx.x;
	//int end = mainArray[i][0] + 1;
	printf("%d:\t%d\n", threadIdx.x, mainArray[i][1]);
	int * startAddy = mainArray[i];
	for(int j = 1; j <= 1; j++)
	{
		//atomicAdd(counts + mainArray[i][j], 1);
	}
}*/

float ** createArr(int size)
{
	float ** arr = (float**)malloc(sizeof(float*) * size);
	for(int i = 0; i < size+1; i++)
	{
		arr[i] = (float*)malloc(sizeof(float) * size + 1);
	}
	return arr;
}

float ** createBlankUserMatrix(UserTableReader r, int columnMax)
{
	float ** arr = (float**)malloc(sizeof(float*) * columnMax+1);

	for (int i = 0; i < r.users.size() +1; i++)
	{
		arr[i] = (float*)malloc(sizeof(float) * columnMax + 1);
	}
	return arr;
}

bool ** createBlankUserDidReviewMatrix(UserTableReader r, int columnMax)
{
	bool ** arr = (bool**)malloc(sizeof(bool*) * columnMax + 1);

	for (int i = 0; i < r.users.size() +1; i++)
	{
		arr[i] = (bool*)malloc(sizeof(bool) * columnMax + 1);
	}
	return arr;
}

void populateUserReviewMatrix(float **userReviewMatrix, bool **originalReviewMatrix, UserTableReader r, MovieReader m)
{
	auto vec = r.users;
	for (auto it = vec.begin(); it != vec.end(); ++it)
	{
		for (auto sit = (*it).ratedMovies.begin(); sit != (*it).ratedMovies.end(); ++sit)
		{
			userReviewMatrix[(*it).userID][m.movieIDMapper[(*sit).movieID]] = (*sit).rating;
		}
	}
}

hipError_t doAlgo()
{
	//int cudaCores = cuda.calculateCores();
	MovieReader m = MovieReader("movie.csv");
	UserTableReader r = UserTableReader("ratings.csv");
	float ** movieMatrix = createArr(m.movieCount);
	float ** userReviewMatrix = createBlankUserMatrix(r, m.movieCount);
	bool ** originalReviewMatrix = createBlankUserDidReviewMatrix(r, m.movieCount);
	populateUserReviewMatrix(userReviewMatrix, originalReviewMatrix, r, m);
	short ** d_movieMatrix;
	short * devptr;
	size_t pitch;
	hipError_t cudaStatus;


	cudaStatus = hipMalloc((void**)&d_movieMatrix, m.movieCount * sizeof(short*));
	hipMallocPitch(&devptr, &pitch, (m.movieCount + 1) * sizeof(short), (m.movieCount + 1));
	hipMemcpy2D(d_movieMatrix, pitch, movieMatrix, (m.movieCount + 1) * sizeof(short), (m.movieCount + 1) * sizeof(short), m.movieCount + 1, hipMemcpyHostToDevice);
 
	//short** temp_d_ptrs = (short **)malloc(sizeof(short*) * mainSize);
	for (int i = 0; i < m.movieCount; i++)
	{
		//hipMalloc((void**)&temp_d_ptrs[i], sizeof(int)* (main[i][0] + 1)); // allocate for 1 int in each int pointer
		//hipMemcpy(temp, main[i], sizeof(int) * getsize, hipMemcpyHostToDevice); // copy data
		//hipMemcpy(devMain + i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}


Error:
	//hipFree();
	//hipFree();
	return cudaStatus;
}



int main()
{

	CUDABackground cuda = CUDABackground();
	doAlgo();
}















/*
hipError_t findFrequents(int* main[], unsigned int *counts, int mainSize, int countsSize)
{
	int **devMain = 0;
	unsigned int * devCounts = 0;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	int** temp_d_ptrs = (int **)malloc(sizeof(int*) * mainSize);
	for (int i = 0; i < mainSize; i++)
	{
		hipMalloc((void**)&temp_d_ptrs[i], sizeof(int)* (main[i][0] + 1)); // allocate for 1 int in each int pointer
		//hipMemcpy(temp, main[i], sizeof(int) * getsize, hipMemcpyHostToDevice); // copy data
		//hipMemcpy(devMain + i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMalloc((void**)&devCounts, countsSize * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devCounts, counts, countsSize * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	findFrequents << <1, 500 >> > (devMain, devCounts);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(counts, devCounts, countsSize * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devMain);
	hipFree(counts);

	return cudaStatus;

}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/