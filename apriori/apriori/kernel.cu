#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "CUDABackground.h"
#include <atomic>
#include <iostream>
#include <pplinterface.h>
#include "FileReader2.h"
#include "UserTableReader.h"
using namespace std;
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t findFrequents(int**main, unsigned int *counts, int mainSize, int countsSize);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void findFrequents(int**mainArray, unsigned int *counts)
{
	int i = threadIdx.x;
	//int end = mainArray[i][0] + 1;
	printf("%d:\t%d\n", threadIdx.x, mainArray[i][1]);
	int * startAddy = mainArray[i];
	for(int j = 1; j <= 1; j++)
	{
		//atomicAdd(counts + mainArray[i][j], 1);
	}
}*/

__global__ void test(float**mainArray)
{
	int i = threadIdx.x / 50;
	int j = threadIdx.x % 50;
	printf("%d:\t%f\n", threadIdx.x, mainArray[1][threadIdx.x]);

}

float ** createArr(int size)
{
	float ** arr = (float**)malloc(sizeof(float*) * size);
	for(int i = 0; i < size+1; i++)
	{
		arr[i] = (float*)malloc(sizeof(float) * size + 1);
	}
	return arr;
}

float ** createBlankUserMatrix(UserTableReader r, int columnMax)
{
	float ** arr = (float**)malloc(sizeof(float*) * columnMax+1);

	for (int i = 0; i < r.users.size() +1; i++)
	{
		arr[i] = (float*)malloc(sizeof(float) * columnMax + 1);
		for (int j = 0; j < r.users.size() + 1; j++)
		{
			arr[i][j] = 0.0;
		}
	}
	return arr;
}

bool ** createBlankUserDidReviewMatrix(UserTableReader r, int columnMax)
{
	bool ** arr = (bool**)malloc(sizeof(bool*) * columnMax + 1);

	for (int i = 0; i < r.users.size() +1; i++)
	{
		arr[i] = (bool*)malloc(sizeof(bool) * columnMax + 1);
		for (int j = 0; j < r.users.size() + 1; j++)
		{
			arr[i][j] = false;
		}
	}
	return arr;
}

void populateUserReviewMatrix(float **userReviewMatrix, bool **originalReviewMatrix, UserTableReader r, MovieReader m)
{
	auto vec = r.users;
	for (auto it = vec.begin(); it != vec.end(); ++it)
	{
		for (auto sit = (*it).ratedMovies.begin(); sit != (*it).ratedMovies.end(); ++sit)
		{
			userReviewMatrix[(*it).userID][m.movieIDMapper[(*sit).movieID]] = (*sit).rating;
			originalReviewMatrix[(*it).userID][m.movieIDMapper[(*sit).movieID]] = true;
		}
	}
}

hipError_t doAlgo()
{
	//int cudaCores = cuda.calculateCores();
	MovieReader m = MovieReader("movie.csv");
	UserTableReader r = UserTableReader("ratings.csv");
	float ** movieMatrix = createArr(m.movieCount);
	float ** userReviewMatrix = createBlankUserMatrix(r, m.movieCount);
	bool ** originalReviewMatrix = createBlankUserDidReviewMatrix(r, m.movieCount);
	populateUserReviewMatrix(userReviewMatrix, originalReviewMatrix, r, m);

	float ** d_movieMatrix;
	float ** d_userReviewMatrix;
	bool ** d_didReviewMatrix;

	float * devptrMovie;
	float * devptrUser;
	bool * devptrUserDidReview;

	size_t pitchMovie;
	size_t pitchUserReview;
	size_t pitchDidReview;

	hipError_t cudaStatus;
	int movieMatrixColumns = m.movieCount + 1;
	int userReviewColumns = m.movieCount + 1;
	int userReviewRows = r.users.size() + 1;
	/*
	float** temp_d_ptrs = (float **)malloc(sizeof(float*) * userReviewRows);
	for (int i = 0; i < userReviewColumns+1; i++)
	{
		hipMalloc((void**)&temp_d_ptrs[i], sizeof(float)* (userReviewColumns)); // allocate for 1 int in each int pointer
		hipMemcpy(temp, main[i], sizeof(int) * getsize, hipMemcpyHostToDevice); // copy data
		//hipMemcpy(devMain + i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}*/

	/*
	cudaStatus = hipMalloc((void**)&d_movieMatrix, movieMatrixColumns * sizeof(float*));
	hipMallocPitch(&devptrMovie, &pitchMovie, movieMatrixColumns * sizeof(float), movieMatrixColumns);
	hipMemcpy2D(d_movieMatrix, pitchMovie, movieMatrix, (m.movieCount + 1) * sizeof(float), (m.movieCount + 1) * sizeof(float), m.movieCount + 1, hipMemcpyHostToDevice);
	*/
	cudaStatus = hipMalloc((void***)&d_userReviewMatrix, userReviewRows * sizeof(float*));
	for (int i = 0; i < userReviewRows; i++)
	{
		float * temp;
		hipMalloc((void**) &(temp), sizeof(float)*userReviewColumns);
		hipMemcpy(temp, userReviewMatrix[i], sizeof(float) * userReviewColumns, hipMemcpyHostToDevice);
		hipMemcpy(d_userReviewMatrix + i, &temp, sizeof(float*), hipMemcpyHostToDevice);
	}

	cudaStatus = hipMalloc((void***)&d_movieMatrix, movieMatrixColumns * sizeof(float*));
	for (int i = 0; i < movieMatrixColumns; i++)
	{
		float * temp;
		hipMalloc((void**) &(temp), sizeof(float)*movieMatrixColumns);
		hipMemcpy(temp, movieMatrix[i], sizeof(float) * movieMatrixColumns, hipMemcpyHostToDevice);
		hipMemcpy(d_movieMatrix + i, &temp, sizeof(float*), hipMemcpyHostToDevice);
	}


	/*hipMallocPitch(&devptrUser, &pitchUserReview, userReviewColumns * sizeof(float), userReviewRows);
	hipMemcpy2D(d_userReviewMatrix, pitchUserReview, userReviewMatrix, userReviewColumns * sizeof(float), userReviewColumns * sizeof(float), userReviewRows, hipMemcpyHostToDevice);
	*/
	test << <1, userReviewRows >> > (d_userReviewMatrix, );


Error:
	//hipFree();
	//hipFree();
	return cudaStatus;
}



int main()
{

	CUDABackground cuda = CUDABackground();
	doAlgo();
}















/*
hipError_t findFrequents(int* main[], unsigned int *counts, int mainSize, int countsSize)
{
	int **devMain = 0;
	unsigned int * devCounts = 0;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc((void**)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	int** temp_d_ptrs = (int **)malloc(sizeof(int*) * mainSize);
	for (int i = 0; i < mainSize; i++)
	{
		hipMalloc((void**)&temp_d_ptrs[i], sizeof(int)* (main[i][0] + 1)); // allocate for 1 int in each int pointer
		//hipMemcpy(temp, main[i], sizeof(int) * getsize, hipMemcpyHostToDevice); // copy data
		//hipMemcpy(devMain + i, &temp, sizeof(int*), hipMemcpyHostToDevice);
	}


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&devMain, mainSize * sizeof(int*));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMalloc((void**)&devCounts, countsSize * sizeof(unsigned int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(devCounts, counts, countsSize * sizeof(unsigned int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	findFrequents << <1, 500 >> > (devMain, devCounts);


	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(counts, devCounts, countsSize * sizeof(unsigned int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(devMain);
	hipFree(counts);

	return cudaStatus;

}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/