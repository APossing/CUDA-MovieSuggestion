#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "CUDABackground.h"
#include <atomic>
#include <iostream>
#include <pplinterface.h>
#include "FileReader2.h"
#include "UserTableReader.h"
#include <chrono>
using namespace std;

__global__ void computeAverageType2(float**mainArray, unsigned short *mainArrayColumns, unsigned short *mainArrayRows)
{
	short column = blockIdx.x * blockDim.x + threadIdx.x + 1;
	float cur;
	if (column < *mainArrayColumns)
	{
		double total = 0;
		unsigned short count = 0;
		for (short i = 1; i < *mainArrayRows; i++)
		{
			cur = mainArray[i][column];
			if (cur >= 0 && cur <= 5)
			{
				total += cur;
				count++;
			}
		}
		mainArray[0][column] = total / count;
		//printf("%d, %d, %f, %d, %f\n", row, 0, total, count, mainArray[row][0]);
	}
}


__global__ void loadTop5(float**userArray, unsigned short *userArrayRows, unsigned short *userArrayColumns, unsigned short *top5UserArray, unsigned short *top5UserArrayColumns, bool **didSelect)
{
	short row = blockIdx.x * blockDim.x + threadIdx.x + 1;
	float biggest;
	short biggestIndex;
	if (row < *userArrayRows)
	{
		for (short i = 0; i < *top5UserArrayColumns; i++)
		{
			for (short j = 1; j < *userArrayColumns; j++)
			{
				if (!didSelect[row][j])
				{
					if (userArray[row][j] > biggest)
					{
						biggest = userArray[row][j];
						biggestIndex = j;
					}
				}
			}
			if (biggestIndex == 0)
				return;
			top5UserArray[row * (*top5UserArrayColumns) + i] = biggestIndex;
			didSelect[row][biggestIndex] = true;
			biggestIndex = 0;
			biggest = 0;
		}
	}
}
__global__ void computeSimularMoviesType2(float**userArray, unsigned short *userArrayRows, float**movieArray, unsigned short *movieArrayColumns)
{
	short movie1 = blockDim.x * blockIdx.x + threadIdx.x + 1;
	short movie2 = blockDim.y * blockIdx.y + threadIdx.y + 1;
	if (movie1 < *movieArrayColumns && movie2 < *movieArrayColumns && movie1 >= movie2)
	{
		//printf("%d,%d\n", movie1, movie2);

		double top = 0;
		float topLeft = 0;
		float topRight = 0;
		double bottomLeft = 0;
		double bottomRight = 0;

		for (short i = 1; i < *userArrayRows; i++)	//for every user
		{
			topLeft = userArray[i][movie1];			//get user rating for movie 1
			if (topLeft < 0 || topLeft > 5)			//if its not filled out by user, set to 0
				topLeft = 0;
			else
			{
				topLeft -= userArray[0][movie1];	//subtracting the average for that movie
			}

			topRight = userArray[i][movie2]; 		//get user rating for movie 2
			if (topRight < 0 || topRight > 5)		//if its not filled out by user, set to 0
				topRight = 0;
			else
			{
				topRight -= userArray[0][movie2]; //subtracting the average for that movie
			}							

			top += topRight * topLeft;				//compute this one and add to sum

			bottomLeft += topLeft * topLeft;		//A^2 and add to A's sum
			bottomRight += topRight * topRight;		//B^2 and add to B's sum
		}
		if (movie1 == 1867 && movie2 == 1)
			printf("");
		//if (movie1 < 11 && movie2 < 11)
			//printf("(%d,%d):\t%f, %f, %f %f\n", movie1, movie2, sqrt(bottomLeft), sqrt(bottomRight), top, top / (sqrt(bottomLeft) * sqrt(bottomRight)));
		float temp = top / (sqrt(bottomLeft) * sqrt(bottomRight));
		movieArray[movie1][movie2] = temp;
		movieArray[movie2][movie1] = temp;
	}

}

//void quicksort(float)

__global__ void computeRecommendedMovies(float**userArray, unsigned short *userArrayColumns, unsigned short *userArrayRows, float**movieArray, bool **didSelect)
{
	short movie = blockDim.x * blockIdx.x + threadIdx.x + 1;
	short user = blockDim.y * blockIdx.y + threadIdx.y + 1;
	float tempSim;
	short selected = 0;
	float top5[6];
	short top5Index[6];
	if (movie < *userArrayColumns && user < *userArrayRows && !didSelect[user][movie])
	{
		for (int i = 1; i < *userArrayColumns; i++)
		{
			if (didSelect[user][i])
			{
				tempSim = movieArray[movie][i];
				if (selected < 5)
				{
					top5[5-selected] = tempSim;
					top5Index[5-selected] = i;
					selected++;
				}
				else
				{
					top5[0] = tempSim;
					top5Index[0] = i;
					float temp;
					short temp2;

					//bubble sort......
					for (int i2 = 0; i2 <= 5; i2++)
					{
						for (int j = 0; j < 5; j++)
						{
							if (top5[j] > top5[j + 1])
							{
								temp = top5[j];
								temp2 = top5Index[j];

								top5[j] = top5[j + 1];
								top5Index[j] = top5Index[j + 1];

								top5[j + 1] = temp;
								top5Index[j + 1] = temp2;
							}
							else if (top5[j] == top5[j + 1] && top5Index[j] > top5Index[j+1])
							{
								temp = top5[j];
								temp2 = top5Index[j];

								top5[j] = top5[j + 1];
								top5Index[j] = top5Index[j + 1];

								top5[j + 1] = temp;
								top5Index[j + 1] = temp2;
							}
						}
					}
				}
			}
		}
		double sum;
		if (movie < 10 && user == 1)
			printf("\n");
		for (int i = 1; i <=selected; i++)
			sum+= top5[i] * movieArray[movie][top5Index[i]];
		if (movie < 10 && user == 1)
			printf("(user,movie,sum,selected,sum/selected)->(%d,%d,%f,%d,%f)\n", user, movie, sum, selected, sum / selected);
		userArray[user][movie] = sum / selected;
	}

}
float ** createArr(int size)
{
	float ** arr = (float**)malloc(sizeof(float*) * size);
	for(int i = 0; i < size+1; i++)
	{
		arr[i] = (float*)malloc(sizeof(float) * size + 1);
		for (int j = 0; j < size + 1; j++)
		{
			arr[i][j] = 0;
		}
	}
	return arr;
}

float ** createBlankUserMatrix(UserTableReader r, int columnMax)
{
	float ** arr = (float**)malloc(sizeof(float*) * columnMax+1);

	for (int i = 0; i < r.users.size() +1; i++)
	{
		arr[i] = (float*)malloc(sizeof(float) * columnMax + 1);
		for (int j = 0; j < r.users.size() + 1; j++)
		{
			arr[i][j] = 6.0;
		}
	}
	return arr;
}

bool ** createBlankUserDidReviewMatrix(UserTableReader r, int columnMax)
{
	bool ** arr = (bool**)malloc(sizeof(bool*) * columnMax + 1);

	for (int i = 0; i < r.users.size() +1; i++)
	{
		arr[i] = (bool*)malloc(sizeof(bool) * columnMax + 1);
		for (int j = 0; j < r.users.size() + 1; j++)
		{
			arr[i][j] = false;
		}
	}
	return arr;
}

void populateUserReviewMatrix(float **userReviewMatrix, bool **originalReviewMatrix, UserTableReader r, MovieReader m)
{
	auto vec = r.users;
	for (auto it = vec.begin(); it != vec.end(); ++it)
	{
		for (auto sit = (*it).ratedMovies.begin(); sit != (*it).ratedMovies.end(); ++sit)
		{
			userReviewMatrix[(*it).userID][m.movieIDMapper[(*sit).movieID]] = (*sit).rating;
			originalReviewMatrix[(*it).userID][m.movieIDMapper[(*sit).movieID]] = true;
		}
	}
}

hipError_t doAlgo()
{
	printf("----------------------StartedCode-----------------------\n");
	auto t1 = std::chrono::high_resolution_clock::now();
	//int cudaCores = cuda.calculateCores();
	MovieReader m = MovieReader("movie.csv");
	UserTableReader r = UserTableReader("ratings.csv");

	auto t2 = std::chrono::high_resolution_clock::now();
	printf("-------Filing Reading completed in %d milliseconds------\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count());
	printf("--------------StartedMatrixBuilding---------------------\n");
	auto t3 = std::chrono::high_resolution_clock::now();


	float ** movieMatrix = createArr(m.movieCount);
	float ** userReviewMatrix = createBlankUserMatrix(r, m.movieCount);
	bool ** originalReviewMatrix = createBlankUserDidReviewMatrix(r, m.movieCount);
	unsigned short * recomendedMoviesMatrix = (unsigned short*)malloc(sizeof(unsigned short)* (r.users.size() + 1) * 5);
	populateUserReviewMatrix(userReviewMatrix, originalReviewMatrix, r, m);


	auto t4 = std::chrono::high_resolution_clock::now();
	printf("-------matrix created completed in %d milliseconds------\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t4 - t3).count());
	printf("-----------------Started Cuda data copy-----------------\n");
	auto t5 = std::chrono::high_resolution_clock::now();


	float ** d_movieMatrix;
	float ** d_userReviewMatrix;
	bool ** d_didReviewMatrix;
	unsigned short * d_recomendedMoviesMatrix;
	string str2;
	hipError_t cudaStatus;
	int movieMatrixColumns = m.movieCount + 1;
	int userReviewColumns = m.movieCount + 1;
	int userReviewRows = r.users.size() + 1;
	unsigned short recomendedMoviesMatrixColumns = 5;
	unsigned short recomendedMoviesMatrixRows = (r.users.size() + 1);


	hipMalloc((void**)&d_recomendedMoviesMatrix, sizeof(unsigned short) * recomendedMoviesMatrixRows * recomendedMoviesMatrixColumns);
	cudaStatus = hipMemcpy(d_recomendedMoviesMatrix, recomendedMoviesMatrix, sizeof(unsigned short)* recomendedMoviesMatrixRows * recomendedMoviesMatrixColumns, hipMemcpyHostToDevice);


	unsigned short * d_recMoviesColumns;
	hipMalloc((void**)&d_recMoviesColumns, sizeof(unsigned short));
	cudaStatus = hipMemcpy(d_recMoviesColumns, &recomendedMoviesMatrixColumns, sizeof(unsigned short), hipMemcpyHostToDevice);


	unsigned short * d_userReviewMatrixColumns;
	hipMalloc((void**)&d_userReviewMatrixColumns, sizeof(unsigned short) * 1);
	cudaStatus = hipMemcpy(d_userReviewMatrixColumns,&userReviewColumns,sizeof(unsigned short), hipMemcpyHostToDevice);


	unsigned short * d_userReviewMatrixRows;
	hipMalloc((void**)&d_userReviewMatrixRows, sizeof(unsigned short) * 1);
	cudaStatus = hipMemcpy(d_userReviewMatrixRows, &userReviewRows, sizeof(unsigned short), hipMemcpyHostToDevice);


	cudaStatus = hipMalloc((void***)&d_userReviewMatrix, userReviewRows * sizeof(float*));
	for (int i = 0; i < userReviewRows; i++)
	{
		float * temp;
		hipMalloc((void**) &(temp), sizeof(float)*userReviewColumns);
		hipMemcpy(temp, userReviewMatrix[i], sizeof(float) * userReviewColumns, hipMemcpyHostToDevice);
		hipMemcpy(d_userReviewMatrix + i, &temp, sizeof(float*), hipMemcpyHostToDevice); 
	}

	cudaStatus = hipMalloc((void***)&d_movieMatrix, movieMatrixColumns * sizeof(float*));
	for (int i = 0; i < movieMatrixColumns; i++)
	{
		float * temp;
		hipMalloc((void**) &(temp), sizeof(float)*movieMatrixColumns);
		hipMemcpy(temp, movieMatrix[i], sizeof(float) * movieMatrixColumns, hipMemcpyHostToDevice);
		hipMemcpy(d_movieMatrix + i, &temp, sizeof(float*), hipMemcpyHostToDevice);
	}

	cudaStatus = hipMalloc((void***)&d_didReviewMatrix, movieMatrixColumns * sizeof(bool*));
	for (int i = 0; i < userReviewRows; i++)
	{
		bool * temp;
		hipMalloc((void**) &(temp), sizeof(bool)*userReviewColumns);
		hipMemcpy(temp, originalReviewMatrix[i], sizeof(bool) * userReviewColumns, hipMemcpyHostToDevice);
		hipMemcpy(d_didReviewMatrix + i, &temp, sizeof(bool*), hipMemcpyHostToDevice);
	}



	int blockX = ceil(userReviewRows / 256.0);
	int blockY = ceil(userReviewRows / 16.0);
	int blockXType2 = ceil(userReviewColumns / 256);

	auto t6 = std::chrono::high_resolution_clock::now();
	printf("-------cuda data copy completed in %d milliseconds------\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t6 - t5).count());
	printf("--------Started Compute Averages for movies-------------\n");
	auto t7 = std::chrono::high_resolution_clock::now();

	computeAverageType2 << <blockXType2, 256 >> > (d_userReviewMatrix, d_userReviewMatrixColumns, d_userReviewMatrixRows);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	auto t8 = std::chrono::high_resolution_clock::now();
	printf("-------Compute Averages for movies completed in %d milliseconds------\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t8 - t7).count());
	printf("--------Started compute simularMovies-------------\n");
	auto t9 = std::chrono::high_resolution_clock::now();

	blockX = ceil(movieMatrixColumns / 16.0);
	blockY = ceil(movieMatrixColumns / 16.0);


	computeSimularMoviesType2<<<dim3(blockX, blockY), dim3(16,16) >>>(d_userReviewMatrix, d_userReviewMatrixRows, d_movieMatrix, d_userReviewMatrixColumns);
	cudaStatus = hipGetLastError();
	if (hipSuccess != hipGetLastError())
		printf("Error!\n");
	cudaStatus = hipGetLastError();
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	blockX = ceil(movieMatrixColumns / 8.0);
	blockY = ceil(userReviewRows / 8.0);

	t8 = std::chrono::high_resolution_clock::now();
	printf("Compute simular movies completed in %d milliseconds\n\n\n", std::chrono::duration_cast<std::chrono::milliseconds>(t8 - t9).count());
	printf("------Started compute recommended movies-----------\n");
	t9 = std::chrono::high_resolution_clock::now();


	computeRecommendedMovies<<<dim3(blockX, blockY), dim3(8, 8) >>>(d_userReviewMatrix, d_userReviewMatrixColumns, d_userReviewMatrixRows, d_movieMatrix, d_didReviewMatrix);
	cudaStatus = hipGetLastError();
	if (hipSuccess != hipGetLastError())
		printf("Error!\n");

	cudaStatus = hipGetLastError();
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	t8 = std::chrono::high_resolution_clock::now();
	printf("Compute recommended movies completed in %d milliseconds\n", std::chrono::duration_cast<std::chrono::milliseconds>(t8 - t9).count());
	blockX = ceil(userReviewRows / 8.0);
	hipError_t cuda3 = hipGetLastError();
	str2 = hipGetErrorString(cuda3);

	loadTop5 << <blockX, 16 >> > (d_userReviewMatrix, d_userReviewMatrixRows, d_userReviewMatrixColumns, d_recomendedMoviesMatrix, d_recMoviesColumns, d_didReviewMatrix);
	hipError_t cuda2 = hipGetLastError();
	str2 = hipGetErrorString(cuda2);
	hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	cudaStatus = hipMemcpy(recomendedMoviesMatrix, d_recomendedMoviesMatrix, sizeof(unsigned short)* recomendedMoviesMatrixRows * recomendedMoviesMatrixColumns, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Cuda MemCpy failed!!\n", cudaStatus);
		goto Error;
	}

	printf("%d, %d, %d, %d", recomendedMoviesMatrix[5], recomendedMoviesMatrix[6], recomendedMoviesMatrix[7], recomendedMoviesMatrix[8]);
Error:
	for (int i = 0; i < movieMatrixColumns; i++)
	{
		hipFree(d_movieMatrix+i);
	}
	for (int i = 0; i < userReviewRows; i++)
	{
		hipFree(d_userReviewMatrix + i);
		hipFree(d_didReviewMatrix + i);
	}
	hipFree(d_didReviewMatrix);
	hipFree(d_movieMatrix);
	hipFree(d_userReviewMatrix);
	hipFree(d_userReviewMatrixColumns);
	hipFree(d_userReviewMatrixRows);

	return cudaStatus;
}



int main()
{

	CUDABackground cuda = CUDABackground();
	doAlgo();
}